#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

using namespace std;

template <typename T>
struct VectorS
{
    bool GH = false;
    T Array[10];
};

template <typename T>
struct PredicateM
{
    __host__ __device__ bool operator()(const VectorS<T> &x) const
    {
        return x.GH;
    };
};

int main()
{

    thrust::host_vector<VectorS<double>> ARR(10);
    ARR[0].GH = true;
    ARR[5].GH = true;

    int result = thrust::count_if(ARR.begin(), ARR.end(), PredicateM<double>());
    cout << result << endl;
    return 0;
};